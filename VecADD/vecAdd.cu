/*-----This is a vector addition--*/
/*---- @ Cuda/c ------*/
/*---- __NS__Bologna__2020__*/



#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void vectorAdd(int* a, int* b, int* c, int n){
// calculate index thread
   int tid = blockIdx.x * blockDim.x + threadIdx.x;
// Make sure we stay in-bounds 
   if(tid < n)
// Vector add
     c[tid] = a[tid] + b[tid];
}

int main()
{
// Number of elements 
    int n = 1 << 20; 

// Host Pointers 
    int* h_a;
    int* h_b;
    int* h_c;

// Device Pointers 
    int* d_a;
    int* d_b;
    int* d_c;
    
    size_t bytes = n * sizeof(int);

//Allocate memoary on host
    h_a = (int*)malloc(bytes);
    h_b = (int*)malloc(bytes);
    h_c = (int*)malloc(bytes);

	for(int i = 0; i <n; i++){
		h_a[i] = 1;
		h_b[i] = 2;  
	}
//Allocate memoary on device 
   hipMalloc(&d_a,bytes);
   hipMalloc(&d_b,bytes);
   hipMalloc(&d_c,bytes);	    

//init block and grid size
   int block_size = 1024;
   int grid_size  = (int)ceil((float) n /block_size); 
   printf("Grid size is %d\n", grid_size);

   hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
   hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice);
    
   vectorAdd<<<grid_size, block_size>>>(d_a, d_b, d_c, n);

   hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);

   for(int i = 0; i < n; i++){
      if(h_c[i] !=3){
        printf("ERROR!\n");
        break;
      }
   }
   printf("Completed Successfully!\n");
   
   free(h_a);
   free(h_b);
   free(h_c);

   hipFree(d_a);
   hipFree(d_b);
   hipFree(d_c);
   return 0;
}



